//Referred Dr.Swenson's Sample code and Nvidia PDF for some code syntaxes and Excerpts. File read logic reference taken from online sources
//like geeks for geeks and cplusplus.com. 

/*
   Akshaya Nagarajan
   ECE 6122 P2
GTID: 903319262
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>


#define T_P_B 1024  //Threads per block for cuda kernel


//struct to store values from conf file

struct init_values {

	std::string dimension;
	float k;
	int timesteps, width, height, depth;
	float default_temp;
	std::vector<int> heatsource;
	std::vector<float> fixed_temp;

};

struct init_values init;


//Logic to read conf file 

void init_readconf(std::string conf)
{
	std::ifstream conf_file(conf.c_str());
	std::string line;
	std::vector<std::string> conf_vector;

	while (std::getline(conf_file, line)) {
		conf_vector.push_back(line);
	}

	std::vector<std::string> temp;
	std::vector<int> temp2;
	std::vector<float> temp3;

	for (int i = 0; i < conf_vector.size(); ++i)
	{
		if ((conf_vector[i].empty() == 0) && (conf_vector[i].find("#") != 0))
		{
			temp.push_back(conf_vector[i]);
		}

	}


	//initialize values non comma separated

	std::stringstream dim0(temp[0]);	
	dim0 >> init.dimension;
	std::stringstream dim1(temp[1]);
	dim1 >> init.k;
	std::stringstream dim2(temp[2]);
	dim2 >> init.timesteps;
	std::stringstream dim3(temp[4]);
	dim3 >> init.default_temp;

	//initialize values comma separated
	if (init.dimension == "2D")
	{
		std::stringstream dim4;
		dim4 << temp[3];
		int a;

		//First get the total grid size
		while(dim4 >> a)
		{
			if (dim4.peek() == ',')
			{
				dim4.ignore();
			}

			temp2.push_back(a);

		}

		init.width = temp2[0];
		init.height = temp2[1];

		for (int i = 5; i < temp.size(); ++i)
		{
			std::stringstream dim5;
			dim5 << temp[i];
			float b;

			//Get the heat source values
			while(dim5 >> b)
			{

				if (dim5.peek() == ',')
				{
					dim5.ignore();
				}

				temp3.push_back(b);

			}
		}


		//Put heat source values into vectors
		for (int i = 0; i < temp3.size(); ++i)
		{	

			init.heatsource.push_back(temp3[i]);

		}



		for (int i = 4; i < temp3.size(); i= i+5)
		{
			init.fixed_temp.push_back(temp3[i]);

		}

		for (int i = 4; i < init.heatsource.size(); i= i+4)
		{
			init.heatsource.erase(init.heatsource.begin() + i);
		}

	}

	else {

		std::stringstream dim4;
		dim4 << temp[3];
		int a;

		//First get the total grid size
		while(dim4 >> a)
		{
			if (dim4.peek() == ',')
			{
				dim4.ignore();
			}

			temp2.push_back(a);

		}

		init.width = temp2[0];
		init.height = temp2[1];
		init.depth = temp2[2];

		for (int i = 5; i < temp.size(); ++i)
		{
			std::stringstream dim5;
			dim5 << temp[i];
			float b;

			//Get the heat source values
			while(dim5 >> b)
			{

				if (dim5.peek() == ',')
				{
					dim5.ignore();
				}

				temp3.push_back(b);

			}
		}

		//Put heat source values into vectors
		for (int i = 0; i < temp3.size(); ++i)
		{	

			init.heatsource.push_back(temp3[i]);

		}

		for (int i = 6; i < temp3.size(); i= i+7)
		{
			init.fixed_temp.push_back(temp3[i]);

		}

		for (int i = 6; i < init.heatsource.size(); i= i+6)
		{
			init.heatsource.erase(init.heatsource.begin() + i);
		}

	}


}



//kernel function for 2D

__global__ void twodfunc(float *arraymain, float *arraytemp, float *arraybool, float k, int width, int height, int N) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x; //Get thread Ids


	if (idx < N)
	{

		float top = arraymain[idx + width];
		float bottom = arraymain[idx - width];
		float left = arraymain[idx -1];
		float right = arraymain[idx +1];

		//Heat Diffusion formula for 8 corner and 1 general case in 2D
		//for 1st element
		if (idx == 0)
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + top + arraymain[idx] + right - 4*arraymain[idx]));

		}

		//for last element 
		else if (idx == (width*height -1))
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + bottom + arraymain[idx] + left - 4*arraymain[idx]));

		}

		//for leftcorner top
		else if ((idx + width == width*height) && (idx%width == 0))
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + bottom + arraymain[idx] + right - 4*arraymain[idx]));

		}

		//for rightcorner bottom
		else if ((idx - width < 0) && (idx%width == (width-1)))
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + top + arraymain[idx] + left - 4*arraymain[idx]));

		}

		//for top 
		else if (idx + width > width*height)
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + bottom + left + right - 4*arraymain[idx]));

		}

		//for bottom 
		else if (idx - width < 0) 
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + top + left + right - 4*arraymain[idx]));

		}

		//for left
		else if (idx%width == 0)
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + top + bottom + right - 4*arraymain[idx]));

		}

		//for right
		else if (idx%width == (width-1))
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(arraymain[idx] + top + left + bottom - 4*arraymain[idx]));

		}


		//general cases
		else 
		{
			arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(top + bottom + left + right - 4*arraymain[idx]));

		}

	}

}


//kernel function for 3D
__global__ void threedfunc(float *arraymain, float *arraytemp, float *arraybool, float k, int width, int height, int depth, int N) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x; //Get thread Ids

	if (idx < N)
	{

		//Setting by default to its own values for corner cases

		float top = arraymain[idx];
		float bottom = arraymain[idx];
		float left = arraymain[idx];
		float right = arraymain[idx];                  
		float front = arraymain[idx]; 
		float back = arraymain[idx];



		//index computation for non corner cases (in order to avoid many loops covering the individual cases)
		int index;
		//for top
		index = idx + width*depth;

		if (index < N)
		{	

			top = arraymain[index];
		}

		//for bottom
		index = idx - width*depth;

		if (index >= 0)
		{

			bottom = arraymain[index];
		}

		//for front
		index = idx%(width*depth);
		index = index/width;

		if (index != 0)
		{

			front = arraymain[idx - width];
		}

		//for back
		if (index != (depth-1))
		{

			back = arraymain[idx + width];
		}

		//for left
		index = idx%width;

		if (index != 0)
		{

			left = arraymain[idx-1];
		}

		//for right
		if (index != width-1)
		{

			right = arraymain[idx+1];
		}

		//general formula for heat diffusion 3D with calculated indexes
		arraytemp[idx] = arraymain[idx] + arraybool[idx]*(k*(front + back + top + bottom + left + right - 6*arraymain[idx]));


	}

}



//main 

int main(int argc, char* argv[]) {

	init_readconf(argv[1]);

	//For 2D
	if (init.dimension == "2D")
	{

		char *filename = (char *)"heatOutput.csv";
		FILE *fp;
		fp = fopen(filename, "w");

		float size = (init.width*init.height) * sizeof(float);
		int N = init.width*init.height;


		float a[N], b[N], c[N];
		float *d_a, *d_b, *d_c;

		for (int i = 0; i < N; ++i)
		{	
			a[i] = init.default_temp;
			b[i] = 0;
			c[i] = 1;
		}


		//logic for conf file and array integration 2D
		int index = 0;

		for (int i = 0; i < init.heatsource.size(); i=i+4)
		{
			for (int j = init.heatsource[i+1]; j < init.heatsource[i+1]+init.heatsource[i+3]; j++)
			{


				for (int k = init.heatsource[i]; k < init.heatsource[i]+init.heatsource[i+2]; k++)
				{

					a[j*init.width + k] = init.fixed_temp[index];
					c[j*init.width + k] = 0;
				}
			}

			index++;

		}

		//initialize array for device 
		hipMalloc((void **)&d_a, size);
		hipMalloc((void **)&d_b, size);
		hipMalloc((void **)&d_c, size);

		//copy to device from host
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

		float *swap;

		//Looping timesteps number of times to get the final grid
		for (int i = 0; i < init.timesteps; i++)
		{
			twodfunc<<<(N + T_P_B-1) / T_P_B, T_P_B>>>(d_a, d_b, d_c, init.k, init.width, init.height, N); //Call kernel function
			hipDeviceSynchronize();
			swap = d_a;
			d_a = d_b;
			d_b = swap;	

		}

		//copy to host from device
		hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
		hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);


		// //Print the Final Grid to CSV file
		for(int i = 0; i < N; i++) {

			if(i!=N-1 && i%init.width == init.width-1 && i != 0)
				fprintf(fp, "%f\n", a[i]);
			else if ((i==0) || (i!=N-1 && i%init.width !=init.width-1))
				fprintf(fp, "%f, ", a[i]);
			else fprintf(fp, "%f\n", a[i]);

		}

		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		fclose(fp);

	}


	//For 3D
	else
	{
		char *filename = (char *)"heatOutput.csv";
		FILE *fp;
		fp = fopen(filename, "w");

		float size = (init.width*init.height*init.depth) * sizeof(float);
		int N = init.width*init.height*init.depth;


		float a[N], b[N], c[N];
		float *d_a, *d_b, *d_c;

		for (int i = 0; i < N; ++i)
		{	
			a[i] = init.default_temp;
			b[i] = 0;
			c[i] = 1;
		}


		//logic for conf file and array integration 3D
		int index = 0;

		for (int i = 0; i < init.heatsource.size(); i=i+6)
		{
			for (int p = init.heatsource[i+1]; p < init.heatsource[i+1]+init.heatsource[i+4]; p++)

			{		
				for (int k = init.heatsource[i+2]; k < init.heatsource[i+2]+init.heatsource[i+5]; k++)

				{
					for (int j = init.heatsource[i]; j < init.heatsource[i]+init.heatsource[i+3]; j++)
					{

						a[p*init.width*init.depth + j + k*init.width] = init.fixed_temp[index];
						c[p*init.width*init.depth + j + k*init.width] = 0;

					}


				}

			}
			index++;
		}


		//initialize array for device 
		hipMalloc((void **)&d_a, size);
		hipMalloc((void **)&d_b, size);
		hipMalloc((void **)&d_c, size);

		//copy to device from host
		hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

		float *swap;

		//Looping timesteps number of times to get the final grid
		for (int i = 0; i < init.timesteps; i++)
		{
			threedfunc<<<(N + T_P_B-1) / T_P_B, T_P_B>>>(d_a, d_b, d_c, init.k, init.width, init.height, init.depth, N);  //Call kernel function
			hipDeviceSynchronize();
			swap = d_a;
			d_a = d_b;
			d_b = swap;	

		}

		//copy to host from device
		hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
		hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);


		//Print the Final Grid to CSV file
		for (int k = 0; k < init.depth; k++)

		{		
			for (int p = 0; p < init.height; p++)

			{
				for (int j = 0; j < init.width; j++)
				{
					if (j == init.width-1)
						fprintf(fp, "%f\n", a[p*init.width*init.depth + j + k*init.width]);
					else fprintf(fp, "%f, ", a[p*init.width*init.depth + j + k*init.width]);

				}


			}
			fprintf(fp, "\n");
		}

		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);
		fclose(fp);
	}

	return 0;
}
